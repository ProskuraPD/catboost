#include "hip/hip_runtime.h"
#include "compute_scores.cuh"

#include <catboost/cuda/methods/kernel/score_calcers.cuh>
#include <catboost/cuda/cuda_util/kernel/instructions.cuh>
#include <catboost/cuda/cuda_util/kernel/random_gen.cuh>
#include <catboost/cuda/cuda_util/kernel/kernel_helpers.cuh>
#include <catboost/cuda/cuda_util/kernel/fill.cuh>
#include <catboost/cuda/cuda_lib/kernel/arch.cuh>
#include <contrib/libs/cub/cub/block/block_reduce.cuh>

#include <cmath>
#include <exception>
#include <cfloat>


namespace NKernel {

//    histId * binFeatureCount * statCount + statId * binFeatureCount + features->FirstFoldIndex;

    template <int BlockSize,
              class TScoreCalcer>
    __global__ void ComputeOptimalSplits(const TCBinFeature* bf,
                                         ui32 binFeatureCount,
                                         const float* histograms,
                                         const double* partStats, int statCount, const ui32* partIds, int pCount,
                                         bool multiclassOptimization,
                                         TScoreCalcer calcer,
                                         TBestSplitProperties* result) {

        float bestScore = FLT_MAX;
        int bestIndex = -1;
        int tid = threadIdx.x;

        result += blockIdx.x + blockIdx.y * gridDim.x;
        partIds += blockIdx.y * pCount;

        for (int offset = blockIdx.x * BlockSize; offset < binFeatureCount; offset += BlockSize * gridDim.x) {
            const int binFeatureId = offset + tid;

            if (binFeatureId >= binFeatureCount) {
                break;
            }
            calcer.NextFeature(bf[binFeatureId]);

            for (int i = 0; i < pCount; i++) {
                const int leafId = __ldg(partIds + i);

                const float weightLeft = max(__ldg(histograms + leafId * statCount * binFeatureCount + binFeatureId), 0.0f);
                const float weightRight = max(__ldg(partStats + leafId * statCount) - weightLeft, 0.0f);

                double totalSumLeft = 0;
                double totalSumPart = 0;
                for (int statId = 1; statId < statCount; ++statId) {
                    float sumLeft = __ldg(histograms + leafId * statCount * binFeatureCount + statId * binFeatureCount + binFeatureId);
                    double partStat = __ldg(partStats + leafId * statCount + statId);
                    totalSumPart += partStat;
                    float sumRight = static_cast<float>(partStat - sumLeft);

                    calcer.AddLeaf(sumLeft, weightLeft);
                    calcer.AddLeaf(sumRight, weightRight);
                    totalSumLeft += sumLeft;


                }
                if (multiclassOptimization) {
                    double totalSumRight = totalSumPart - totalSumLeft;
                    calcer.AddLeaf(-totalSumLeft, weightLeft);
                    calcer.AddLeaf(-totalSumRight, weightRight);
                }
            }

            const float score = calcer.GetScore();


            if (score < bestScore) {
                bestScore = score;
                bestIndex = binFeatureId;
            }
        }


        __shared__ float scores[BlockSize];
        scores[tid] = bestScore;

        __shared__ int indices[BlockSize];
        indices[tid] = bestIndex;
        __syncthreads();

        for (ui32 s = BlockSize >> 1; s > 0; s >>= 1) {
            if (tid < s) {
                if (scores[tid] > scores[tid + s] || (scores[tid] == scores[tid + s] && indices[tid] > indices[tid + s]) ) {
                    scores[tid] = scores[tid + s];
                    indices[tid] = indices[tid + s];
                }
            }
            __syncthreads();
        }

        if (!tid) {
            const int index = indices[0];

            if (index != -1 && index < binFeatureCount) {
                result->FeatureId = bf[index].FeatureId;
                result->BinId = bf[index].BinId;
                result->Score = scores[0];
            } else {
                result->FeatureId = static_cast<ui32>(-1);
                result->BinId = static_cast<ui32>(-1);
                result->Score = FLT_MAX;

            }
        }
    }




    void ComputeOptimalSplits(const TCBinFeature* binaryFeatures, ui32 binaryFeatureCount,
                              const float* histograms,
                              const double* partStats, int statCount,
                              ui32* partIds, int partBlockSize, int partBlockCount,
                              TBestSplitProperties* result, ui32 argmaxBlockCount,
                              EScoreFunction scoreFunction,
                              bool multiclassOptimization,
                              double l2,
                              bool normalize,
                              double scoreStdDev,
                              ui64 seed,
                              TCudaStream stream) {
        const int blockSize = 128;

        dim3 numBlocks;
        numBlocks.x = argmaxBlockCount;
        numBlocks.y = partBlockCount;
        numBlocks.z = 1;

        #define RUN() \
        ComputeOptimalSplits<blockSize, TScoreCalcer> << < numBlocks, blockSize, 0, stream >> > (binaryFeatures, binaryFeatureCount, histograms, partStats,  statCount, partIds, partBlockSize, multiclassOptimization, scoreCalcer, result);


        switch (scoreFunction)
        {
            case  EScoreFunction::SolarL2: {
                using TScoreCalcer = TSolarScoreCalcer;
                TScoreCalcer scoreCalcer(static_cast<float>(l2));
                RUN()
                break;
            }
            case  EScoreFunction::SatL2: {
                using TScoreCalcer = TSatL2ScoreCalcer;
                TScoreCalcer scoreCalcer(static_cast<float>(l2));
                RUN()
                break;
            }
            case  EScoreFunction::LOOL2: {
                using TScoreCalcer = TLOOL2ScoreCalcer;
                TScoreCalcer scoreCalcer(static_cast<float>(l2));
                RUN()
                break;
            }
            case EScoreFunction::L2:
            case EScoreFunction::NewtonL2: {
                using TScoreCalcer = TL2ScoreCalcer;
                TScoreCalcer scoreCalcer(static_cast<float>(l2));
                RUN()
                break;
            }
            case  EScoreFunction::Correlation:
            case  EScoreFunction::NewtonCorrelation: {
                using TScoreCalcer = TCorrelationScoreCalcer;
                TCorrelationScoreCalcer scoreCalcer(static_cast<float>(l2),
                                                    normalize,
                                                    static_cast<float>(scoreStdDev),
                                                    seed);
                RUN()
                break;
            }
            default: {
                throw std::exception();
            }
        }
        #undef RUN
    }


    template <int BlockSize>
    __global__ void ComputeTargetVarianceImpl(const float* stats,
                                              ui32 size,
                                              ui32 statCount,
                                              ui64 statLineSize,
                                              bool isMulticlass,
                                              double* aggregatedStats) {

        ui32 i = BlockSize * blockIdx.x + threadIdx.x;

        float weightedSum = 0;
        float weightedSum2 = 0;
        float totalWeight = 0;

        while (i < size) {
            const float w = stats[i];
            if (w > 1e-15f) {
                float statSum = 0;
                for (ui32 statId = 1; statId < statCount; ++statId) {
                    const float wt = stats[i + statLineSize * statId];
                    weightedSum += wt;
                    weightedSum2 += wt * wt / w; //cause we need sum w * t * t
                    statSum += wt;
                }
                if (isMulticlass) {
                    weightedSum += -statSum;
                    weightedSum2 += statSum * statSum / w;
                }
                totalWeight += w;
            }
            i += gridDim.x * BlockSize;
        }

        using BlockReduce = typename hipcub::BlockReduce<double, BlockSize>;
        __shared__ typename BlockReduce::TempStorage tempStorage;

        double blockWeightedSum = weightedSum;
        blockWeightedSum = BlockReduce(tempStorage).Sum(blockWeightedSum);

        double blockWeightedSum2 = weightedSum2;


        blockWeightedSum2 = BlockReduce(tempStorage).Sum(blockWeightedSum2);

        double blockTotalWeight = totalWeight;
        blockTotalWeight = BlockReduce(tempStorage).Sum(blockTotalWeight);



        if (threadIdx.x == 0) {
            TAtomicAdd<double>::Add(aggregatedStats, blockWeightedSum);
            TAtomicAdd<double>::Add(aggregatedStats + 1, blockWeightedSum2);
            TAtomicAdd<double>::Add(aggregatedStats + 2, blockTotalWeight);
        }
    }


    void ComputeTargetVariance(const float* stats,
                               ui32 size,
                               ui32 statCount,
                               ui64 statLineSize,
                               bool isMulticlass,
                               double* aggregatedStats,
                               TCudaStream stream) {

        const ui32 blockSize = 512;
        const ui32 numBlocks = min(4 * TArchProps::SMCount(), CeilDivide(size, blockSize));
        FillBuffer(aggregatedStats, 0.0, 3, stream);
        if (numBlocks) {
            ComputeTargetVarianceImpl<blockSize><<<numBlocks, blockSize, 0, stream>>>(stats, size, statCount, statLineSize, isMulticlass, aggregatedStats);
        }
    }


}
